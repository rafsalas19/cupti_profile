#include <cupti_target.h>
#include <cupti_profiler_target.h>
#include <nvperf_host.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <string>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "cuptiErrorCheck.h"
#include "cuptiMetrics.h"


using namespace std;
string slist[] {"achieved_occupancy",
"branch_efficiency",
"cf_executed",
"double_precision_fu_utilization",
"dram_read_bytes",
"dram_read_throughput",
"dram_read_transactions",
"dram_utilization",
"dram_write_bytes",
"dram_write_throughput",
"dram_write_transactions",
"eligible_warps_per_cycle",
"flop_count_dp_add",
"flop_count_dp_fma",
"flop_count_dp_mul",
"flop_count_hp_add",
"flop_count_hp_fma",
"flop_count_hp_mul",
"flop_count_sp_add",
"flop_count_sp_fma",
"flop_count_sp_mul",
"flop_dp_efficiency",
"flop_hp_efficiency",
"flop_sp_efficiency",
"gld_efficiency",
"gld_throughput",
"gld_transactions",
"gld_transactions_per_request",
"global_atomic_requests",
"global_load_requests",
"global_reduction_requests",
"global_store_requests",
"gst_efficiency",
"gst_throughput",
"gst_transactions",
"gst_transactions_per_request",
"half_precision_fu_utilization",
"inst_bit_convert",
"inst_compute_ld_st",
"inst_control",
"inst_executed",
"inst_executed_global_atomics",
"inst_executed_global_loads",
"inst_executed_global_reductions",
"inst_executed_global_stores",
"inst_executed_local_loads",
"inst_executed_local_stores",
"inst_executed_shared_loads",
"inst_executed_shared_stores",
"inst_executed_surface_atomics",
"inst_executed_surface_reductions",
"inst_executed_surface_stores",
"inst_executed_tex_ops",
"inst_fp_16",
"inst_fp_32",
"inst_fp_64",
"inst_integer",
"inst_inter_thread_communication",
"inst_issued",
"inst_misc",
"inst_per_warp",
"ipc",
"issue_slot_utilization",
"issue_slots",
"issued_ipc",
"l1_sm_lg_utilization",
"l2_atomic_throughput",
"l2_atomic_transactions",
"l2_global_atomic_store_bytes",
"l2_global_load_bytes",
"l2_local_load_bytes",
"l2_read_throughput",
"l2_read_transactions",
"l2_surface_load_bytes",
"l2_surface_store_bytes",
"l2_tex_hit_rate",
"l2_tex_read_hit_rate",
"l2_tex_read_throughput",
"l2_tex_read_transactions",
"l2_tex_write_hit_rate",
"l2_tex_write_throughput",
"l2_tex_write_transactions",
"l2_utilization",
"l2_write_throughput",
"l2_write_transactions",
"ldst_fu_utilization",
"local_load_requests",
"local_load_throughput",
"local_load_transactions",
"local_load_transactions_per_request",
"local_store_requests",
"local_store_throughput",
"local_store_transactions",
"local_store_transactions_per_request",
"pcie_total_data_received",
"pcie_total_data_transmitted",
"shared_efficiency",
"shared_load_throughput",
"shared_load_transactions",
"shared_store_throughput",
"shared_store_transactions",
"shared_utilization",
"single_precision_fu_utilization",
"sm_efficiency",
"sm_tex_utilization",
"special_fu_utilization",
"stall_constant_memory_dependency",
"stall_inst_fetch",
"stall_memory_dependency",
"stall_not_selected",
"stall_sleeping",
"stall_texture",
"surface_atomic_requests",
"surface_load_requests",
"surface_reduction_requests",
"surface_store_requests",
"sysmem_read_bytes",
"sysmem_read_throughput",
"sysmem_read_transactions",
"sysmem_write_bytes",
"sysmem_write_throughput",
"sysmem_write_transactions",
"tensor_precision_fu_utilization",
"tex_cache_hit_rate",
"tex_fu_utilization",
"tex_sm_tex_utilization",
"tex_sm_utilization",
"texture_load_requests",
"warp_execution_efficiency",
"warp_nonpred_execution_efficiency"};





int main(int argc, char* argv[])
{
	//ProfileSession ps;
	
	//ps.addMetric("met");
	for(int i=0;i<130;i++){
		cout<<"{"<<1000+i<<",\""<<slist[i]<<"\"},"<<endl;
	}
	cout<<"done"<<endl;
	return 0;
}

